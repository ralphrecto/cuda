// Parallelized Conway's Game of Life
#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <hip/hip_runtime.h>
#include <iostream>

#define GRID_WIDTH   64
#define GRID_HEIGHT  64
#define SEED         1337
#define BLOCK_WIDTH  32
#define BLOCK_HEIGHT 32

// grid helpers

// - representation: 1 cell == 1 unsigned char
// - TODO: pack this more densely (1 bit/cell instead of 1 byte/cell)
// - first/last row/col are shadows -- copies of wraparound. so actual cells
//   start at (1, 1) for example.
typedef unsigned char* gol_grid;

__host__ __device__ static int block_index(int x, int y) {
    return y * BLOCK_WIDTH + x;
}

__host__ __device__ static int grid_index(int x, int y) {
    return y * GRID_WIDTH + x;
}

// set grid to 1/0 at x/y position; handle shadow rows/cols
__host__ __device__ void set(gol_grid grid, unsigned int x, unsigned int y, bool live) {
    int i = grid_index(x, y);
    grid[i] = (unsigned char) (live ? 1 : 0);
}

// indicates whether cell at (x, y) is alive
__host__ __device__ bool is_live(gol_grid grid, unsigned int x, unsigned int y) {
    int i = grid_index(x, y);
    return grid[i] > 0;
}

// computes byte size of entire grid
size_t get_grid_size(unsigned int width, unsigned int height) {
    // pad w/ shadow/wraparound cells
    unsigned int num_cells = (width + 2) * (height + 2);
    return (size_t) ceil(num_cells / sizeof(unsigned char));
}

// allocate + randomly initialize the grid
gol_grid create_grid() {
    size_t grid_size = get_grid_size(GRID_WIDTH, GRID_HEIGHT);
    gol_grid grid = (gol_grid) malloc(grid_size);

    // initialize non-shadow cells
    for (int x = 1; x < GRID_WIDTH + 1; x++) {
        for (int y = 1; y < GRID_HEIGHT + 1; y++) {
            int i = grid_index(x, y);
            grid[i] = (unsigned char) (rand() % sizeof(unsigned char));
        }
    }

    // initialize shadow rows
    for (int x = 1; x < GRID_WIDTH + 1; x++) {
        int i_first = grid_index(x, 0);
        int i_first_shadow = grid_index(x, GRID_HEIGHT - 1);

        int i_last = grid_index(x, GRID_HEIGHT);
        int i_last_shadow = grid_index(x, 1);

        grid[i_first] = grid[i_first_shadow];
        grid[i_last] = grid[i_last_shadow];
    }

    // initialize shadow columns
    for (int y = 1; y < GRID_HEIGHT + 1; y++) {
        int i_first = grid_index(0, y);
        int i_first_shadow = grid_index(GRID_WIDTH - 1, y);

        int i_last = grid_index(GRID_WIDTH, y);
        int i_last_shadow = grid_index(1, y);

        grid[i_first] = grid[i_first_shadow];
        grid[i_last] = grid[i_last_shadow];
    }

    return grid;
}

// kernel code

__global__ void gol_kernel(
    gol_grid grid,
    gol_grid next_grid
) {
    extern __shared__ unsigned char block_grid[];

    // logical coords (start at (0, 0))
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    int y = blockDim.y * blockIdx.y + threadIdx.y;

    if (x >= GRID_WIDTH || y >= GRID_HEIGHT) return;

    // mapped to grid coords, taking shadow rows/cols into account
    int grid_x = x+1;
    int grid_y = y+1;

    // initialize block grid in shared mem
    for (int dx = -1; dx < 2; dx++) {
        for (int dy = -1; dy < 2; dy++) {
            int xi = grid_x + dx;
            int yi = grid_y + dy;

            if (xi < 0 || xi >= GRID_WIDTH || yi < 0 || yi >= GRID_HEIGHT) continue;

            // TODO remove redundant reads/writes of overlaps
            // TODO double check this pointer math
            int i = grid_index(xi, yi);
            int block_i = block_index(xi % BLOCK_WIDTH, yi % BLOCK_HEIGHT);

            block_grid[block_i] = grid[i];
        }
    }

    __syncthreads();

    // compute state for tick
    int live_neighbors = 0;
    for (int xi = grid_x - 1; xi < grid_x + 2 && xi <= GRID_WIDTH && xi >= 0; xi++) {
        for (int yi = grid_y - 1; yi < grid_y + 2 && yi <= GRID_HEIGHT && yi >= 0; yi++) {
            if (xi == grid_x && yi == grid_y) continue;

            int block_i = block_index(xi % BLOCK_WIDTH, yi % BLOCK_HEIGHT);
            if (block_grid[block_i] > 0) {
                live_neighbors += 1;
            }
        }
    }

    bool currently_alive = block_grid[block_index(grid_x % BLOCK_WIDTH, grid_y % BLOCK_HEIGHT)];
    bool next_alive = false;
    if (currently_alive) {
        if (live_neighbors == 2 || live_neighbors == 3) {
            next_alive = true;
        } else {
            next_alive = false;
        }
    } else if (live_neighbors == 3) {
        next_alive = true;
    }

    set(next_grid, grid_x, grid_y, next_alive);
}

void checkCudaError(const char* message) {
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        std::cerr << "CUDA Error after " << message << ": " << hipGetErrorName(error) << "; " << hipGetErrorString(error) << std::endl;
        exit(EXIT_FAILURE);
    }
}

int main(int argc, char* argv[]) {
    int num_iter;
    if (argc != 2) {
        printf("%d [num iterations]\n");
        exit(1);
    } else {
        num_iter = atoi(argv[1]);
    }

    srand(SEED);

    // initialize state
    size_t grid_size = get_grid_size(GRID_WIDTH, GRID_HEIGHT);
    gol_grid h_grid = create_grid();

    gol_grid d_grid;
    gol_grid d_grid_next;

    hipMalloc((void**) &d_grid, grid_size);
    hipMalloc((void**) &d_grid_next, grid_size);
    hipMemcpy(d_grid, h_grid, grid_size, hipMemcpyHostToDevice);
    hipMemcpy(d_grid_next, h_grid, grid_size, hipMemcpyHostToDevice);

    dim3 grid_dim(GRID_WIDTH / BLOCK_WIDTH, GRID_HEIGHT / BLOCK_HEIGHT);
    dim3 block_dim(BLOCK_WIDTH, BLOCK_HEIGHT);
    size_t block_size = (BLOCK_WIDTH + 2) * (BLOCK_HEIGHT + 2) * sizeof(unsigned char);

    printf("starting game of life sim...\n");
    printf("\x1b[2J");

    for (int i = 0; i < num_iter; i++) {
        gol_kernel<<<grid_dim, block_dim, block_size>>>(d_grid, d_grid_next);
        hipDeviceSynchronize();
        checkCudaError("game of life kernel");

        gol_grid d_temp = d_grid;
        d_grid = d_grid_next;
        d_grid_next = d_temp;

        if (i % 100 == 0 && i > 0 || i == 0) {
            printf("num iterations: %d\n", i);

            hipMemcpy(h_grid, d_grid, grid_size, hipMemcpyDeviceToHost);
            for (int x = 1; x < GRID_WIDTH + 1; x++) {
                for (int y = 1; y < GRID_HEIGHT + 1; y++) {
                    int i = grid_index(x, y);
                    if (h_grid[i] > 0) {
                        printf("x");
                    } else {
                        printf("%d", h_grid[i], i);
                    }
                }
                printf("\n");
            }
        }
    }

    printf("done!\n");
    printf("total num iterations: %d\n", num_iter);

    free(h_grid);
    hipFree(d_grid);

    return 0;
}