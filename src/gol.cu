#include "hip/hip_runtime.h"
// Parallelized Conway's Game of Life
#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <hip/hip_runtime.h>

#define GRID_WIDTH   640
#define GRID_HEIGHT  480
#define SEED         1337

// grid helpers

// representation: 1 cell == 1 unsigned char
// TODO: pack this more densely (1 bit/cell instead of 1 byte/cell)
typedef unsigned char* gol_grid;

static int grid_index(unsigned int x, unsigned int y) {
    return y * GRID_WIDTH + x;
}

// set grid to 1/0 at x/y position; handle shadow rows/cols
void set(gol_grid grid, unsigned int x, unsigned int y, bool live) {
    int i = grid_index(x, y);
    grid[i] = (unsigned char) (live ? 1 : 0);
}

bool is_live(gol_grid grid, unsigned int x, unsigned int y) {
    int i = grid_index(x, y);
    return grid[i] == 0;
}

size_t get_grid_size(unsigned int width, unsigned int height) {
    // pad w/ shadow/wraparound cells
    unsigned int num_cells = (width + 2) * (height + 2);
    return (size_t) ceil(num_cells / sizeof(unsigned char));
}

// allocate + randomly initialize the grid
gol_grid create_grid() {
    size_t grid_size = get_grid_size(GRID_WIDTH, GRID_HEIGHT);
    gol_grid grid = (gol_grid) malloc(grid_size);

    // initialize non-shadow cells
    for (int x = 1; x < GRID_WIDTH + 1; x++) {
        for (int y = 1; y < GRID_HEIGHT + 1; y++) {
            int i = grid_index(x, y);
            grid[i] = (unsigned char) (rand() % sizeof(unsigned char));
        }
    }

    // initialize shadow rows
    for (int x = 1; x < GRID_WIDTH + 1; x++) {
        int i_first = grid_index(x, 0);
        int i_first_shadow = grid_index(x, GRID_HEIGHT - 1);

        int i_last = grid_index(x, GRID_HEIGHT);
        int i_last_shadow = grid_index(x, 1);

        grid[i_first] = grid[i_first_shadow];
        grid[i_last] = grid[i_last_shadow];
    }

    // initialize shadow columns
    for (int y = 1; y < GRID_HEIGHT + 1; y++) {
        int i_first = grid_index(0, y);
        int i_first_shadow = grid_index(GRID_WIDTH - 1, y);

        int i_last = grid_index(GRID_WIDTH, y);
        int i_last_shadow = grid_index(1, y);

        grid[i_first] = grid[i_first_shadow];
        grid[i_last] = grid[i_last_shadow];
    }

    return grid;
}

// kernel code

__global__ void gol_kernel() {
    extern __shared__ gol_grid block_grid;

    int x = blockDim.x * blockIdx.x + threadIdx.x;
    int y = blockDim.y * blockIdx.y + threadIdx.y;

    if (x >= GRID_WIDTH || y >= GRID_HEIGHT) return;
}

int main() {
    srand(SEED);

    // initialize state
    size_t grid_size = get_grid_size(GRID_WIDTH, GRID_HEIGHT);
    gol_grid h_grid = create_grid();

    gol_grid d_grid;
    hipMalloc((void**) &d_grid, grid_size);
    hipMemcpy(d_grid, h_grid, grid_size, hipMemcpyHostToDevice);

    free(h_grid);
    hipFree(d_grid);
}