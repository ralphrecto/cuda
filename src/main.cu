#include "stdio.h"
#include "hip/hip_runtime.h"

#define THREADS_PER_BLOCK 256

__global__ void vector_add_kernel(float* a, float* b, float* c, int n) {
  int i = blockDim.x*blockIdx.x + threadIdx.x;
  if (i < n) {
    c[i] = a[i] + b[i];
  }
}

void vector_add(float* a, float* b, float* c, int n) {
  float* d_a;
  float* d_b;
  float* d_c;

  size_t size = (size_t) n * sizeof(float);

  hipMalloc((void**) &d_a, size);
  hipMalloc((void**) &d_b, size);
  hipMalloc((void**) &d_c, size);

  hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
  hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
  hipMemcpy(d_c, c, size, hipMemcpyHostToDevice);

  vector_add_kernel<<<ceil(n / (1.0 * THREADS_PER_BLOCK)), THREADS_PER_BLOCK>>>(d_a, d_b, d_c, n);

  hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);
}

typedef float (*mapf)(float);

__device__ float incr(float a) {
  return a + 1.0;
}

__device__ mapf d_incr = incr;

__global__ void map_kernel(float* a, float* b, float (*func)(float)) {
  int i = blockDim.x*blockIdx.x + threadIdx.x;
  b[i] = func(a[i]);
}

void map(float* a, float* b, int n, float (*func)(float)) {
  float* d_a;
  float* d_b;
  size_t size = (size_t) n * sizeof(float);

  hipMalloc((void**) &d_a, size);
  hipMalloc((void**) &d_b, size);

  hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);

  map_kernel<<<ceil(n / (1.0 * THREADS_PER_BLOCK)), THREADS_PER_BLOCK>>>(d_a, d_b, func);

  hipMemcpy(b, d_b, size, hipMemcpyDeviceToHost);

  hipFree(d_a);
  hipFree(d_b);
}

int main() {
  float a[4] = {1, 1, 1, 1};
  float b[4] = {1, 2, 1, 0};
  float* c = new float[4];

  mapf h_incr;
  hipMemcpyFromSymbol(&h_incr, HIP_SYMBOL(d_incr), sizeof(mapf));

  map(a, c, 4, h_incr);
  printf("%f %f %f %f", c[0], c[1], c[2], c[3]);
}
