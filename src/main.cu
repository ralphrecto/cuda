#include "stdio.h"
#include "hip/hip_runtime.h"

#define THREADS_PER_BLOCK 256

__global__ void vector_add_kernel(float* a, float* b, float* c, int n) {
  int i = blockDim.x*blockIdx.x + threadIdx.x;
  if (i < n) {
    c[i] = a[i] + b[i];
  }
}

void vector_add(float* a, float* b, float* c, int n) {
  float* d_a;
  float* d_b;
  float* d_c;

  size_t size = (size_t) n * sizeof(float);

  hipMalloc((void**) &d_a, size);
  hipMalloc((void**) &d_b, size);
  hipMalloc((void**) &d_c, size);

  hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
  hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
  hipMemcpy(d_c, c, size, hipMemcpyHostToDevice);

  vector_add_kernel<<<ceil(n / (1.0 * THREADS_PER_BLOCK)), THREADS_PER_BLOCK>>>(d_a, d_b, d_c, n);

  hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);
}

int main() {
  float a[4] = {1, 1, 1, 1};
  float b[4] = {1, 2, 1, 0};
  float* c = new float[4];

  vector_add(a, b, c, 4);
  printf("%f %f %f %f", c[0], c[1], c[2], c[3]);
}
