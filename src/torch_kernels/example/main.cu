#include "hip/hip_runtime.h"
// NOTE: This is a copy of cuda_extension_kernel.cu. It's kept here to test
// collision handling when a C++ file and CUDA file share the same filename.
// Setuptools can't deal with this at all, so the setup.py-based test uses
// cuda_extension_kernel.cu and the JIT test uses this file. Symlinks don't
// work well on Windows, so this is the most thorough solution right now.

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <c10/cuda/CUDAException.h>

#include <ATen/ATen.h>

__global__ void sigmoid_add_kernel(
    const float* __restrict__ x,
    const float* __restrict__ y,
    float* __restrict__ output,
    const int size) {
  const int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < size) {
    const float sigmoid_x = 1.0f / (1.0f + __expf(-x[index]));
    const float sigmoid_y = 1.0f / (1.0f + __expf(-y[index]));
    output[index] = sigmoid_x + sigmoid_y;
  }
}

void sigmoid_add_cuda(const float* x, const float* y, float* output, int size) {
  const int threads = 1024;
  const int blocks = (size + threads - 1) / threads;
  sigmoid_add_kernel<<<blocks, threads>>>(x, y, output, size);
  C10_CUDA_KERNEL_LAUNCH_CHECK();
}

__global__ void sum_kernel(
    const float* __restrict__ x,
    float* __restrict__ output,
    const int size) {
  const int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < size) {
    atomicAdd(output, x[index]);
  }
}

void sum_cuda(const float* x, float* output, int size) {
  const int threads = 1024;
  const int blocks = (size + threads - 1) / threads;
  sum_kernel<<<blocks, threads>>>(x, output, size);
  C10_CUDA_KERNEL_LAUNCH_CHECK();
}

__global__ void sum_fast_kernel(
    const float* __restrict__ x,
    float* __restrict__ output,
    const int size) {
  const int tid = threadIdx.x;
  const int index = blockIdx.x * blockDim.x + tid;
  __shared__ float s_x[1024];

  if (tid >= 1024) {
    return;
  }

  s_x[tid] = x[index];
  __syncthreads();

  for (int stride = blockDim.x >> 1; stride > 0; stride >>= 1) {
    if (tid < stride) {
      s_x[tid] += s_x[tid + stride];
    } 

    __syncthreads();
  }

  if (tid == 0) atomicAdd(output, s_x[0]);
}

void sum_fast_cuda(const float* x, float* output, int size) {
  const int threads = 1024;
  const int blocks = (size + threads - 1) / threads;
  sum_kernel<<<blocks, threads>>>(x, output, size);
  C10_CUDA_KERNEL_LAUNCH_CHECK();
}